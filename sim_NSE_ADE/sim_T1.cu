#include "hip/hip_runtime.h"
//#define AB_PATTERN

#include "lbm3d/core.h"
#include "lbm3d/d3q7/eq.h"
#include "lbm3d/d3q7/col_srt.h"
#include "lbm3d/d3q7/col_mrt.h"
#include "lbm3d/d3q7/col_clbm.h"
// exactly one streaming header must be included
#ifdef AA_PATTERN
	#include "lbm3d/d3q7/streaming_AA.h"
#endif
#ifdef AB_PATTERN
	#include "lbm3d/d3q7/streaming_AB.h"
#endif
#include "lbm3d/d3q7/bc.h"
#include "lbm3d/d3q7/macro.h"
#include "lbm3d/state_NSE_ADE.h"

template < typename TRAITS >
struct NSE_Data_FreeRhoConstInflow : NSE_Data< TRAITS >
{
	using dreal = typename TRAITS::dreal;
	using idx = typename TRAITS::idx;

	dreal inflow_vx = 0;
	dreal inflow_vy = 0;
	dreal inflow_vz = 0;

	template < typename LBM_KS >
	CUDA_HOSTDEV void inflow(LBM_KS &KS, idx x, idx y, idx z)
	{
		KS.vx  = inflow_vx;
		KS.vy  = inflow_vy;
		KS.vz  = inflow_vz;
	}
};

#if 0
template < typename NSE >
#ifdef USE_CUDA
__global__ void cudaLBMComputeQCriterion(
	typename NSE::DATA SD,
	short int rank,
	short int nproc
)
#else
void LBMComputeQCriterion(
	typename NSE::DATA SD,
	typename NSE::TRAITS::idx x,
	typename NSE::TRAITS::idx y,
	typename NSE::TRAITS::idx z,
	short int rank,
	short int nproc
)
#endif
{
	using dreal = typename NSE::TRAITS::dreal;
	using idx = typename NSE::TRAITS::idx;
	using map_t = typename NSE::TRAITS::map_t;

	#ifdef USE_CUDA
	idx x = threadIdx.x + blockIdx.x * blockDim.x;
	idx y = threadIdx.y + blockIdx.y * blockDim.y;
	idx z = threadIdx.z + blockIdx.z * blockDim.z;
	#endif
	map_t gi_map = SD.map(x, y, z);

	idx xp,xm,yp,ym,zp,zm;
	if (NSE::BC::isPeriodic(gi_map))
	{
		// handle overlaps between GPUs
//		xp = (!SD.overlap_right && x == SD.X-1) ? 0 : (x+1);
//		xm = (!SD.overlap_left && x == 0) ? (SD.X-1) : (x-1);
		xp = (nproc == 1 && x == SD.X()-1) ? 0 : (x+1);
		xm = (nproc == 1 && x == 0) ? (SD.X()-1) : (x-1);
		yp = (y == SD.Y()-1) ? 0 : (y+1);
		ym = (y == 0) ? (SD.Y()-1) : (y-1);
		zp = (z == SD.Z()-1) ? 0 : (z+1);
		zm = (z == 0) ? (SD.Z()-1) : (z-1);
	} else {
		// handle overlaps between GPUs
		// NOTE: ghost layers of lattice sites are assumed in the x-direction, so x+1 and x-1 always work
		xp = x+1;
		xm = x-1;
		yp = MIN(y+1, SD.Y()-1);
		ym = MAX(y-1,0);
		zp = MIN(z+1, SD.Z()-1);
		zm = MAX(z-1,0);
	}

		struct Tensor
		{
			dreal xx=0,xy=0,xz=0;
			dreal yx=0,yy=0,yz=0;
			dreal zx=0,zy=0,zz=0;
		};
		Tensor G;

		// grad vel tensor
		if (y == 0 || z == 0 || y == SD.Y()-1 || z == SD.Z()-1)
		{
			// do nothing
			G.xx = 0;
			G.xy = 0;
			G.xz = 0;
			G.yx = 0;
			G.yy = 0;
			G.yz = 0;
			G.zx = 0;
			G.zy = 0;
			G.zz = 0;
		}
		else if (x == 0) {
			// forward difference for x
			G.xx = SD.macro(NSE::MACRO::e_vx,xp,y,z) - SD.macro(NSE::MACRO::e_vx,x,y,z);
			G.yx = SD.macro(NSE::MACRO::e_vy,xp,y,z) - SD.macro(NSE::MACRO::e_vy,x,y,z);
			G.zx = SD.macro(NSE::MACRO::e_vz,xp,y,z) - SD.macro(NSE::MACRO::e_vz,x,y,z);
			// central differences for y,z
			G.xy = n1o2 * (SD.macro(NSE::MACRO::e_vx,x,yp,z) - SD.macro(NSE::MACRO::e_vx,x,ym,z));
			G.yy = n1o2 * (SD.macro(NSE::MACRO::e_vy,x,yp,z) - SD.macro(NSE::MACRO::e_vy,x,ym,z));
			G.zy = n1o2 * (SD.macro(NSE::MACRO::e_vz,x,yp,z) - SD.macro(NSE::MACRO::e_vz,x,ym,z));
			G.xz = n1o2 * (SD.macro(NSE::MACRO::e_vx,x,y,zp) - SD.macro(NSE::MACRO::e_vx,x,y,zm));
			G.yz = n1o2 * (SD.macro(NSE::MACRO::e_vy,x,y,zp) - SD.macro(NSE::MACRO::e_vy,x,y,zm));
			G.zz = n1o2 * (SD.macro(NSE::MACRO::e_vz,x,y,zp) - SD.macro(NSE::MACRO::e_vz,x,y,zm));
		}
		else if (x == SD.X()-1) {
			// backward difference for x
			G.xx = SD.macro(NSE::MACRO::e_vx,x,y,z) - SD.macro(NSE::MACRO::e_vx,xm,y,z);
			G.yx = SD.macro(NSE::MACRO::e_vy,x,y,z) - SD.macro(NSE::MACRO::e_vy,xm,y,z);
			G.zx = SD.macro(NSE::MACRO::e_vz,x,y,z) - SD.macro(NSE::MACRO::e_vz,xm,y,z);
			// central differences for y,z
			G.xy = n1o2 * (SD.macro(NSE::MACRO::e_vx,x,yp,z) - SD.macro(NSE::MACRO::e_vx,x,ym,z));
			G.yy = n1o2 * (SD.macro(NSE::MACRO::e_vy,x,yp,z) - SD.macro(NSE::MACRO::e_vy,x,ym,z));
			G.zy = n1o2 * (SD.macro(NSE::MACRO::e_vz,x,yp,z) - SD.macro(NSE::MACRO::e_vz,x,ym,z));
			G.xz = n1o2 * (SD.macro(NSE::MACRO::e_vx,x,y,zp) - SD.macro(NSE::MACRO::e_vx,x,y,zm));
			G.yz = n1o2 * (SD.macro(NSE::MACRO::e_vy,x,y,zp) - SD.macro(NSE::MACRO::e_vy,x,y,zm));
			G.zz = n1o2 * (SD.macro(NSE::MACRO::e_vz,x,y,zp) - SD.macro(NSE::MACRO::e_vz,x,y,zm));
		}
		else {
			// central differences
			G.xx = n1o2 * (SD.macro(NSE::MACRO::e_vx,xp,y,z) - SD.macro(NSE::MACRO::e_vx,xm,y,z));
			G.xy = n1o2 * (SD.macro(NSE::MACRO::e_vx,x,yp,z) - SD.macro(NSE::MACRO::e_vx,x,ym,z));
			G.xz = n1o2 * (SD.macro(NSE::MACRO::e_vx,x,y,zp) - SD.macro(NSE::MACRO::e_vx,x,y,zm));
			G.yx = n1o2 * (SD.macro(NSE::MACRO::e_vy,xp,y,z) - SD.macro(NSE::MACRO::e_vy,xm,y,z));
			G.yy = n1o2 * (SD.macro(NSE::MACRO::e_vy,x,yp,z) - SD.macro(NSE::MACRO::e_vy,x,ym,z));
			G.yz = n1o2 * (SD.macro(NSE::MACRO::e_vy,x,y,zp) - SD.macro(NSE::MACRO::e_vy,x,y,zm));
			G.zx = n1o2 * (SD.macro(NSE::MACRO::e_vz,xp,y,z) - SD.macro(NSE::MACRO::e_vz,xm,y,z));
			G.zy = n1o2 * (SD.macro(NSE::MACRO::e_vz,x,yp,z) - SD.macro(NSE::MACRO::e_vz,x,ym,z));
			G.zz = n1o2 * (SD.macro(NSE::MACRO::e_vz,x,y,zp) - SD.macro(NSE::MACRO::e_vz,x,y,zm));
		}

		// q criterion from definition: Q = - sum_ij d_i u_j d_j u_i
		const dreal q = G.xx*G.yy + G.yy*G.zz + G.xx*G.zz - G.zx*G.xz - G.yz*G.zy - G.xy*G.yx;
		SD.macro(NSE::MACRO::e_qcrit,x,y,z) = q;
}

template < typename ADE >
#ifdef USE_CUDA
__global__ void cudaLBMComputePhiGradMag(
	typename ADE::DATA SD,
	short int rank,
	short int nproc
)
#else
void cudaLBMComputePhiGradMag(
	typename ADE::DATA SD,
	typename ADE::TRAITS::idx x,
	typename ADE::TRAITS::idx y,
	typename ADE::TRAITS::idx z,
	short int rank,
	short int nproc
)
#endif
{
	using dreal = typename ADE::TRAITS::dreal;
	using idx = typename ADE::TRAITS::idx;
	using map_t = typename ADE::TRAITS::map_t;

	#ifdef USE_CUDA
	idx x = threadIdx.x + blockIdx.x * blockDim.x;
	idx y = threadIdx.y + blockIdx.y * blockDim.y;
	idx z = threadIdx.z + blockIdx.z * blockDim.z;
	#endif
	map_t gi_map = SD.map(x, y, z);

	idx xp,xm,yp,ym,zp,zm;
	if (ADE::BC::isPeriodic(gi_map))
	{
		// handle overlaps between GPUs
//		xp = (!SD.overlap_right && x == SD.X-1) ? 0 : (x+1);
//		xm = (!SD.overlap_left && x == 0) ? (SD.X-1) : (x-1);
		xp = (nproc == 1 && x == SD.X()-1) ? 0 : (x+1);
		xm = (nproc == 1 && x == 0) ? (SD.X()-1) : (x-1);
		yp = (y == SD.Y()-1) ? 0 : (y+1);
		ym = (y == 0) ? (SD.Y()-1) : (y-1);
		zp = (z == SD.Z()-1) ? 0 : (z+1);
		zm = (z == 0) ? (SD.Z()-1) : (z-1);
	} else {
		// handle overlaps between GPUs
		// NOTE: ghost layers of lattice sites are assumed in the x-direction, so x+1 and x-1 always work
		xp = x+1;
		xm = x-1;
		yp = MIN(y+1, SD.Y()-1);
		ym = MAX(y-1,0);
		zp = MIN(z+1, SD.Z()-1);
		zm = MAX(z-1,0);
	}

		struct Vector
		{
			dreal x=0;
			dreal y=0;
			dreal z=0;
		};
		Vector G;

		// grad phi vector
		if (y == 0 || z == 0 || y == SD.Y()-1 || z == SD.Z()-1)
		{
			// do nothing
			G.x = 0;
			G.y = 0;
			G.z = 0;
		}
		else if (x == 0) {
			// forward difference for x
			G.x = SD.macro(ADE::MACRO::e_phi,xp,y,z) - SD.macro(ADE::MACRO::e_phi,x,y,z);
			G.y = 0;
			G.z = 0;
		}
		else if (x == SD.X()-1) {
			// backward difference for x
			G.x = SD.macro(ADE::MACRO::e_phi,x,y,z) - SD.macro(ADE::MACRO::e_phi,xm,y,z);
			G.y = 0;
			G.z = 0;
		}
		else {
			// central differences
			G.x = n1o2 * (SD.macro(ADE::MACRO::e_phi,xp,y,z) - SD.macro(ADE::MACRO::e_phi,xm,y,z));
			G.y = n1o2 * (SD.macro(ADE::MACRO::e_phi,x,yp,z) - SD.macro(ADE::MACRO::e_phi,x,ym,z));
			G.z = n1o2 * (SD.macro(ADE::MACRO::e_phi,x,y,zp) - SD.macro(ADE::MACRO::e_phi,x,y,zm));
		}

		SD.macro(ADE::MACRO::e_phigradmag2,x,y,z) = G.x*G.x + G.y*G.y + G.z*G.z;
}

template < typename TRAITS >
struct D3Q27_MACRO_QCriterion : D3Q27_MACRO_Base< TRAITS >
{
	using dreal = typename TRAITS::dreal;
	using idx = typename TRAITS::idx;

	enum { e_rho, e_vx, e_vy, e_vz, e_fx, e_fy, e_fz, e_qcrit, N};

	template < typename LBM_DATA, typename LBM_KS >
	CUDA_HOSTDEV static void outputMacro(LBM_DATA &SD, LBM_KS &KS, idx x, idx y, idx z)
	{
		SD.macro(e_rho, x, y, z) = KS.rho;
		SD.macro(e_vx, x, y, z)  = KS.vx;
		SD.macro(e_vy, x, y, z)  = KS.vy;
		SD.macro(e_vz, x, y, z)  = KS.vz;
	}

	template < typename LBM_DATA, typename LBM_KS >
	CUDA_HOSTDEV static void copyQuantities(LBM_DATA &SD, LBM_KS &KS, idx x, idx y, idx z)
	{
		KS.lbmViscosity = SD.lbmViscosity;
		KS.fx = SD.fx;
		KS.fy = SD.fy;
		KS.fz = SD.fz;
	}
};
#endif

// 3D test domain
template < typename NSE, typename ADE >
struct StateLocal : State_NSE_ADE<NSE, ADE>
{
	using TRAITS = typename NSE::TRAITS;
	using BLOCK_NSE = LBM_BLOCK< NSE >;
	using BLOCK_ADE = LBM_BLOCK< ADE >;

	using State<NSE>::nse;
	using State_NSE_ADE<NSE, ADE>::ade;
	using State<NSE>::cnt;
	using State<NSE>::vtk_helper;
	using State<NSE>::log;

	using idx = typename TRAITS::idx;
	using real = typename TRAITS::real;
	using dreal = typename TRAITS::dreal;
	using point_t = typename TRAITS::point_t;
	using lat_t = Lattice<3, real, idx>;

	real lbmInflowDensity = no1;

	// constructor
	StateLocal(const TNL::MPI::Comm& communicator, lat_t ilat, real iphysViscosity, real iphysVelocity, real iphysDt, real iphysDiffusion)
		: State_NSE_ADE<NSE, ADE>(communicator, ilat, iphysViscosity, iphysDt, iphysDiffusion)
	{
		for (auto& block : nse.blocks)
		{
//			block.data.inflow_rho = no1;
			block.data.inflow_vx = nse.phys2lbmVelocity(iphysVelocity);
			block.data.inflow_vy = 0;
			block.data.inflow_vz = 0;
		}

		for (auto& block : ade.blocks)
		{
			// TODO: phys -> lbm conversion for concentration?
			block.data.inflow_phi = 1e-3;
		}
	}

	void setupBoundaries() override
	{
		nse.setBoundaryX(0, NSE::BC::GEO_INFLOW); 		// left
		nse.setBoundaryX(nse.lat.global.x()-1, NSE::BC::GEO_OUTFLOW_EQ);
//		nse.setBoundaryX(nse.lat.global.x()-1, NSE::BC::GEO_OUTFLOW_RIGHT);

		nse.setBoundaryZ(1, NSE::BC::GEO_WALL);		// top
		nse.setBoundaryZ(nse.lat.global.z()-2, NSE::BC::GEO_WALL);	// bottom
		nse.setBoundaryY(1, NSE::BC::GEO_WALL); 		// back
		nse.setBoundaryY(nse.lat.global.y()-2, NSE::BC::GEO_WALL);		// front

		// extra layer needed due to A-A pattern
		nse.setBoundaryZ(0, NSE::BC::GEO_NOTHING);		// top
		nse.setBoundaryZ(nse.lat.global.z()-1, NSE::BC::GEO_NOTHING);	// bottom
		nse.setBoundaryY(0, NSE::BC::GEO_NOTHING); 		// back
		nse.setBoundaryY(nse.lat.global.y()-1, NSE::BC::GEO_NOTHING);		// front

		// ADE boundaries
		ade.setBoundaryX(0, ADE::BC::GEO_INFLOW); 		// left
		ade.setBoundaryX(ade.lat.global.x()-1, ADE::BC::GEO_OUTFLOW_RIGHT);

		ade.setBoundaryZ(1, ADE::BC::GEO_WALL);		// top
		ade.setBoundaryZ(ade.lat.global.z()-2, ADE::BC::GEO_WALL);	// bottom
		ade.setBoundaryY(1, ADE::BC::GEO_WALL); 		// back
		ade.setBoundaryY(ade.lat.global.y()-2, ADE::BC::GEO_WALL);		// front

		// extra layer needed due to A-A pattern
		ade.setBoundaryZ(0, ADE::BC::GEO_NOTHING);		// top
		ade.setBoundaryZ(ade.lat.global.z()-1, ADE::BC::GEO_NOTHING);	// bottom
		ade.setBoundaryY(0, ADE::BC::GEO_NOTHING); 		// back
		ade.setBoundaryY(ade.lat.global.y()-1, ADE::BC::GEO_NOTHING);		// front

		// draw a sphere
		if (1)
		{
			int cy=floor(0.2/nse.lat.physDl);
			int cz=floor(0.2/nse.lat.physDl);
			int cx=floor(0.45/nse.lat.physDl);
			real radius=0.05; // 10 cm diameter
			int range=ceil(radius/nse.lat.physDl)+1;
			for (int py=cy-range;py<=cy+range;py++)
			for (int pz=cz-range;pz<=cz+range;pz++)
			for (int px=cx-range;px<=cx+range;px++)
				//if (NORM( (real)(px-cx)*nse.lat.physDl, (real)(py-cy)*nse.lat.physDl, (real)(pz-cz)*nse.lat.physDl) < radius )
				if ((real)(px-cx)*nse.lat.physDl < radius && (real)(py-cy)*nse.lat.physDl < radius && (real)(pz-cz)*nse.lat.physDl < radius )
				{
					nse.setMap(px,py,pz,NSE::BC::GEO_WALL);
					ade.setMap(px,py,pz,ADE::BC::GEO_WALL);
				}
		}

		// draw a cylinder
		if (0)
		{
			//int cy=floor(0.2/nse.lat.physDl);
			int cz=floor(0.2/nse.lat.physDl);
			int cx=floor(0.45/nse.lat.physDl);
			real radius=0.05; // 10 cm diameter
			int range=ceil(radius/nse.lat.physDl)+1;
			//for (int py=cy-range;py<=cy+range;py++)
			for (int pz=cz-range;pz<=cz+range;pz++)
			for (int px=cx-range;px<=cx+range;px++)
			for (int py=0;py<=nse.lat.global.y()-1;py++)
				if (NORM( (real)(px-cx)*nse.lat.physDl,0, (real)(pz-cz)*nse.lat.physDl) < radius )
				{
					nse.setMap(px,py,pz,NSE::BC::GEO_WALL);
					ade.setMap(px,py,pz,ADE::BC::GEO_WALL);
				}
		}

		// draw a block
		if (0)
		{
			//int cy=floor(0.2/nse.lat.physDl);
			//int cz=floor(0.20/nse.lat.physDl);
			int cx=floor(0.20/nse.lat.physDl);
			//int range=nse.lat.global.z()/4;
			int width=nse.lat.global.z()/10;
			//for (int py=cy-range;py<=cy+range;py++)
			//for (int pz=0;pz<=cz;pz++)
			for (int px=cx;px<=cx+width;px++)
			for (int pz=1;pz<=nse.lat.global.z()-2;pz++)
			for (int py=1;py<=nse.lat.global.y()-2;py++)
				if (!((pz>=nse.lat.global.z()*4/10 &&  pz<=nse.lat.global.z()*6/10) && (py>=nse.lat.global.y()*4/10 && py<=nse.lat.global.y()*6/10)))
				{
					nse.setMap(px,py,pz,NSE::BC::GEO_WALL);
					ade.setMap(px,py,pz,ADE::BC::GEO_WALL);
				}
		}
	}

	void updateKernelVelocities() override
	{
//		for (auto& block : nse.blocks)
//			block.data.inflow_rho = lbmInflowDensity;
	}

#if 0
	void computeBeforeLBMKernel() override
	{
		#ifdef USE_CUDA
		auto get_grid_size = [] (const auto& block, idx x = 0, idx y = 0, idx z = 0) -> dim3
		{
			dim3 gridSize;
			if (x > 0)
				gridSize.x = x;
			else
				gridSize.x = TNL::roundUpDivision(block.local.x(), block.block_size.x());
			if (y > 0)
				gridSize.y = y;
			else
				gridSize.y = TNL::roundUpDivision(block.local.y(), block.block_size.y());
			if (z > 0)
				gridSize.z = z;
			else
				gridSize.z = TNL::roundUpDivision(block.local.z(), block.block_size.z());

			return gridSize;
		};
		#endif

		for (auto& block : nse.blocks)
		{
		#ifdef USE_CUDA
			const dim3 gridSize = get_grid_size(block);
			cudaLBMComputeQCriterion< NSE ><<<gridSize, block.block_size>>>(block.data, nse.rank, nse.nproc);
			hipStreamSynchronize(0);
			checkCudaDevice;
		#else
			#pragma omp parallel for schedule(static) collapse(2)
			for (idx x = 0; x < block.local.x(); x++)
			for (idx z = 0; z < block.local.z(); z++)
			for (idx y = 0; y < block.local.y(); y++)
				LBMComputeQCriterion< NSE >(block.data, nse.rank, nse.nproc, x, y, z);
		#endif
		}

		for (auto& block : ade.blocks)
		{
		#ifdef USE_CUDA
			const dim3 gridSize = get_grid_size(block);
			cudaLBMComputePhiGradMag< ADE ><<<gridSize, block.block_size>>>(block.data, nse.rank, nse.nproc);
			hipStreamSynchronize(0);
			checkCudaDevice;
		#else
			#pragma omp parallel for schedule(static) collapse(2)
			for (idx x = 0; x < block.local.x(); x++)
			for (idx z = 0; z < block.local.z(); z++)
			for (idx y = 0; y < block.local.y(); y++)
				cudaLBMComputePhiGradMag< ADE >(block.data, nse.rank, nse.nproc, x, y, z);
		#endif
		}
	}
#endif

	bool outputData(const BLOCK_NSE& block, int index, int dof, char *desc, idx x, idx y, idx z, real &value, int &dofs) override
	{
		int k=0;
		if (index==k++) return vtk_helper("lbm_density", block.hmacro(NSE::MACRO::e_rho,x,y,z), 1, desc, value, dofs);
		if (index==k++)
		{
			switch (dof)
			{
				case 0: return vtk_helper("velocity", nse.lbm2physVelocity(block.hmacro(NSE::MACRO::e_vx,x,y,z)), 3, desc, value, dofs);
				case 1: return vtk_helper("velocity", nse.lbm2physVelocity(block.hmacro(NSE::MACRO::e_vy,x,y,z)), 3, desc, value, dofs);
				case 2: return vtk_helper("velocity", nse.lbm2physVelocity(block.hmacro(NSE::MACRO::e_vz,x,y,z)), 3, desc, value, dofs);
			}
		}
//		if (index==k++) return vtk_helper("lbm_qcriterion", block.hmacro(NSE::MACRO::e_qcrit,x,y,z), 1, desc, value, dofs);
		return false;
	}

	bool outputData(const BLOCK_ADE& block, int index, int dof, char *desc, idx x, idx y, idx z, real &value, int &dofs) override
	{
		int k=0;
		if (index==k++) return vtk_helper("lbm_phi", block.hmacro(ADE::MACRO::e_phi,x,y,z), 1, desc, value, dofs);
//		if (index==k++) return vtk_helper("lbm_phigradmag2", block.hmacro(ADE::MACRO::e_phigradmag2,x,y,z), 1, desc, value, dofs);
		return false;
	}

	void probe1() override
	{
		if (nse.iterations != 0)
		{
			// inflow density extrapolation
			idx x = 5;
			idx y = nse.lat.global.y()/2;
			idx z = nse.lat.global.z()/2;
			for (auto& block : nse.blocks)
			if (block.isLocalIndex(x, y, z))
			{
				real oldlbmInflowDensity = lbmInflowDensity;
				lbmInflowDensity = block.dmacro.getElement(NSE::MACRO::e_rho, x, y, z);
				log("[probe: lbm inflow density changed from {:e} to {:e}", oldlbmInflowDensity, lbmInflowDensity);
			}
		}
	}
};

template < typename NSE, typename ADE >
int simT1_test(int RESOLUTION = 2)
{
	using idx = typename NSE::TRAITS::idx;
	using real = typename NSE::TRAITS::real;
	using point_t = typename NSE::TRAITS::point_t;
	using lat_t = Lattice<3, real, idx>;

	int block_size=32;
	int X = 128*RESOLUTION;// width in pixels
	//	int Y = 41*RESOLUTION;// height in pixels --- top and bottom walls 1px
	//	int Z = 41*RESOLUTION;// height in pixels --- top and bottom walls 1px
	int Y = block_size*RESOLUTION;// height in pixels --- top and bottom walls 1px
	int Z = Y;// height in pixels --- top and bottom walls 1px
	real LBM_VISCOSITY = 0.001/3.0;//1.0/6.0; /// GIVEN: optimal is 1/6
	real PHYS_HEIGHT = 0.41; // [m] domain height (physical)
	real PHYS_VISCOSITY = 1.552e-5; // [m^2/s] fluid viscosity of air
	real PHYS_VELOCITY = 1.0;
	real PHYS_DL = PHYS_HEIGHT/((real)Y-2);
	real PHYS_DT = LBM_VISCOSITY / PHYS_VISCOSITY*PHYS_DL*PHYS_DL;//PHYS_HEIGHT/(real)LBM_HEIGHT;
	real PHYS_DIFFUSION = 2.552e-05; // [m^2/s] diffusion coeff for the ADE
	point_t PHYS_ORIGIN = {0., 0., 0.};

	// initialize the lattice
	lat_t lat;
	lat.global = typename lat_t::CoordinatesType( X, Y, Z );
	lat.physOrigin = PHYS_ORIGIN;
	lat.physDl = PHYS_DL;

	StateLocal< NSE, ADE > state(MPI_COMM_WORLD, lat, PHYS_VISCOSITY, PHYS_VELOCITY, PHYS_DT, PHYS_DIFFUSION);
	state.setid("sim_T1_res{:02d}_np{:03d}", RESOLUTION, state.nse.nproc);
//	state.printIter = 100;
//	state.printIter = 100;
	state.nse.physFinalTime = 10.0;
	state.cnt[PRINT].period = 0.01;
//	state.cnt[PROBE1].period = 0.001;
	// test
//	state.cnt[PRINT].period = 100*PHYS_DT;
//	state.nse.physFinalTime = 1000*PHYS_DT;
//	state.cnt[VTK3D].period = 1000*PHYS_DT;
//	state.cnt[SAVESTATE].period = 600;  // save state every [period] of wall time
//	state.check_savestate_flag = false;
//	state.wallTime = 60;
	// RCI
//	state.nse.physFinalTime = 0.5;
//	state.cnt[VTK3D].period = 0.5;
//	state.cnt[SAVESTATE].period = 3600;  // save state every [period] of wall time
//	state.check_savestate_flag = false;
//	state.wallTime = 3600 * 23.5;

	// add cuts
	state.cnt[VTK2D].period = 0.01;
	state.add2Dcut_X(X/2,"cutsX/cut_X");
	state.add2Dcut_Y(Y/2,"cutsY/cut_Y");
	state.add2Dcut_Z(Z/2,"cutsZ/cut_Z");

//	state.cnt[VTK3D].period = 0.001;
//	state.cnt[VTK3DCUT].period = 0.001;
//	state.add3Dcut(X/4,Y/4,Z/4, X/2,Y/2,Z/2, 2, "box");

	execute(state);

	return 0;
}

//template < typename TRAITS=TraitsSP >
template < typename TRAITS=TraitsDP >
void run(int RES)
{
	using NSE_COLL = D3Q27_CUM< TRAITS, D3Q27_EQ_INV_CUM<TRAITS> >;
	using NSE_CONFIG = LBM_CONFIG<
				TRAITS,
				D3Q27_KernelStruct,
//				NSE_Data_ConstInflow< TRAITS >,
				// FIXME: FreeRho inflow condition leads to lower velocity in the domain (approx 70%)
				NSE_Data_FreeRhoConstInflow< TRAITS >,
				NSE_COLL,
				typename NSE_COLL::EQ,
				D3Q27_STREAMING< TRAITS >,
				D3Q27_BC_All,
				D3Q27_MACRO_Default< TRAITS >,
//				D3Q27_MACRO_QCriterion< TRAITS >,
				D3Q27_MACRO_Void< TRAITS >
			>;

//	using ADE_COLL = D3Q7_SRT< TRAITS >;
//	using ADE_COLL = D3Q7_MRT< TRAITS >;
	using ADE_COLL = D3Q7_CLBM< TRAITS >;
	using ADE_CONFIG = LBM_CONFIG<
				TRAITS,
				D3Q7_KernelStruct,
				ADE_Data_ConstInflow< TRAITS >,
				ADE_COLL,
				typename ADE_COLL::EQ,
				D3Q7_STREAMING< TRAITS >,
				D3Q7_BC_All,
				D3Q7_MACRO_Default< TRAITS >,
				D3Q7_MACRO_Void< TRAITS >
			>;

	simT1_test< NSE_CONFIG, ADE_CONFIG >(RES);
}

int main(int argc, char **argv)
{
	TNLMPI_INIT mpi(argc, argv);

	const int pars=1;
	if (argc <= pars)
	{
		printf("error: required %d parameters:\n %s res[1,...]\n", pars, argv[0]);
		return 1;
	}
	int res = atoi(argv[1]);
	if (res < 1) { printf("error: res=%d out of bounds [1, ...]\n",res); return 1; }

	run(res);

	return 0;
}
